#include <iostream>
#include <hip/hip_runtime.h>

#define CHK_CUDA(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define ONE_KILOBYTE 1024
#define ONE_MEGABYTE ((1024 * ONE_KILOBYTE))

// TODO: figure out optimization stuff
template<int stride, int elemCount, int loopCount>
__global__ void fixed_pchase(int *base)
{
    for (int loopIdx = 0; loopIdx < loopCount; loopIdx++) {
        for (int i = 0; i < elemCount; i++) {
            base[stride * i] = i;
        }
    }
}

void bench_memory_sequential()
{
    int stride_max = 128 * ONE_MEGABYTE;
    int stride_min = 1;
    int arena_size = stride_max * 4;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *devMem;
    // TODO: test on flags
    hipMallocManaged(&devMem, arena_size);

    CHK_CUDA(hipEventRecord(start));
    fixed_pchase<1, 400000, 1000> <<<1, 1>>>(devMem);
    CHK_CUDA(hipEventRecord(stop));

    // wait until event complete
    CHK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    std::cout << hipGetErrorString(hipDeviceSynchronize()) << std::endl;

    std::cout << "Time elapsed: " << std::to_string(milliseconds) << std::endl;
}

int main() {
    CHK_CUDA(hipSetDevice(0));

    bench_memory_sequential();

    return 0;
}
